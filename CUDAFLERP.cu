#include "hip/hip_runtime.h"
/*******************************************************************
*   CUDAFLERP.cu
*   CUDAFLERP
*
*	Author: Kareem Omar
*	kareem.omar@uah.edu
*	https://github.com/komrad36
*
*	Last updated Jan 7, 2017
*******************************************************************/
//
// The file CUDAFLERP.h exposes two extremely high performance GPU
// resize operations,
// CUDAFLERP (bilinear interpolation), and 
// CUDAFNERP (nearest neighbor interpolation), for 32-bit float
// grayscale data.
//
// For 8-bit unsigned integer data, see the CUDALERP project instead.
//
// CUDAFLERP offers superior accuracy to CUDA's built-in texture
// interpolator at comparable performance. The accuracy if compiled
// with -use-fast-math off is nearly equivalent to my CPU interpolator,
// KLERP, while still being as fast as the built-in interpolation.
// 
// Particularly for large images, CUDAFLERP dramatically outperforms
// even the highly tuned CPU AVX2 versions.
// 
// All functionality is contained in the header 'CUDAFLERP.h' and
// the source file 'CUDAFLERP.cu' and has no external dependencies at all.
// 
// The file 'main.cpp' is an example and speed test driver.
//

#include "CUDAFLERP.h"

__global__ void
#ifndef __INTELLISENSE__
__launch_bounds__(256, 0)
#endif
CUDAFNERP_kernel(const hipTextureObject_t d_img_tex, const float gxs, const float gys, float* __restrict const d_out, const int neww) {
	uint32_t x = (blockIdx.x << 9) + (threadIdx.x << 1);
	const uint32_t y = blockIdx.y;
	const float fy = y*gys;
#pragma unroll
	for (int i = 0; i < 2; ++i, ++x) {
		const float fx = x*gxs;
		const float res = tex2D<float>(d_img_tex, fx, fy);
		if (x < neww) d_out[y*neww + x] = res;
	}
}

__global__ void
#ifndef __INTELLISENSE__
__launch_bounds__(256, 0)
#endif
CUDAFLERP_kernel(const hipTextureObject_t d_img_tex, const float gxs, const float gys, float* __restrict const d_out, const int neww) {
	uint32_t x = (blockIdx.x << 9) + (threadIdx.x << 1);
	const uint32_t y = blockIdx.y;
	const float fy = (y + 0.5f)*gys - 0.5f;
	const float wt_y = fy - floor(fy);
	const float invwt_y = 1.0f - wt_y;
#pragma unroll
	for (int i = 0; i < 2; ++i, ++x) {
		const float fx = (x + 0.5f)*gxs - 0.5f;
		const float4 f = tex2Dgather<float4>(d_img_tex, fx + 0.5f, fy + 0.5f);
		const float wt_x = fx - floor(fx);
		const float invwt_x = 1.0f - wt_x;
		const float xa = invwt_x*f.w + wt_x*f.z;
		const float xb = invwt_x*f.x + wt_x*f.y;
		const float res = invwt_y*xa + wt_y*xb;
		if (x < neww) d_out[y*neww + x] = res;
	}
}

void CUDAFNERP(const hipTextureObject_t d_img_tex, const int oldw, const int oldh, float* __restrict const d_out, const uint32_t neww, const uint32_t newh) {
	const float gxs = static_cast<float>(oldw) / static_cast<float>(neww);
	const float gys = static_cast<float>(oldh) / static_cast<float>(newh);
	CUDAFNERP_kernel<<<{((neww - 1) >> 9) + 1, newh}, 256>>>(d_img_tex, gxs, gys, d_out, neww);
	hipDeviceSynchronize();
}

void CUDAFLERP(const hipTextureObject_t d_img_tex, const int oldw, const int oldh, float* __restrict const d_out, const uint32_t neww, const uint32_t newh) {
	const float gxs = static_cast<float>(oldw) / static_cast<float>(neww);
	const float gys = static_cast<float>(oldh) / static_cast<float>(newh);
	CUDAFLERP_kernel<<<{((neww - 1) >> 9) + 1, newh}, 256>>>(d_img_tex, gxs, gys, d_out, neww);
	hipDeviceSynchronize();
}
